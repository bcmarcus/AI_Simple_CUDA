#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include <cmath>
#include <unistd.h>

#include <coreutils/classes/matrixes/Matrix3D.cuh>
#include <coreutils/functions/debug/print.hpp>
#include <coreutils/util/time.hpp>
#include <coreutils/util/cudaErrors.cuh>

#include "../layers/BasicLayer.cuh"
#include "../weights/BasicWeight.cuh"

using namespace std;
using namespace coreutils::classes::matrixes;
using namespace coreutils::functions::debug;
using namespace artificialIntelligence::classes;
using namespace artificialIntelligence::functions::activation;

#define MAX_BLOCK_SIZE 8192

BasicLayer::BasicLayer (Matrix3D* layerMatrix, Matrix3D* biasMatrix, BasicWeight* weights, ActivationType activationType) {
   this->layerMatrixes = new Matrix3D* [1];
	this->layerMatrixes[0] = layerMatrix;

	this->biasMatrixes = new Matrix3D*[1];
	this->biasMatrixes[0] = nullptr;
   if (biasMatrix != nullptr) {
      this->biasMatrixes[0] = new Matrix3D(biasMatrix->getLength(), biasMatrix->getWidth(), biasMatrix->getHeight());
      this->getBias()->setMatrix(biasMatrix);
   }

	this->weights = (WeightBase**) new BasicWeight*[1];
   this->weights[0] = weights;

   this->next = (LayerBase**) new BasicLayer*[1];
	this->next[0] = nullptr;
   this->prev = (LayerBase**) new BasicLayer*[1];
	this->prev[0] = nullptr;

	this->layerMatrixCount = 1;
	this->biasCount = biasMatrix != nullptr;
	this->weightsCount = weights != nullptr;
	this->nextCount = 0;
	this->prevCount = 0;

	this->type = LayerBase::LayerType::Basic;
	this->activationType = activationType;
}


BasicLayer::BasicLayer (int length, int width, int height, ActivationType activationType) {
	this->layerMatrixes = new Matrix3D* [1];
   this->layerMatrixes[0] = new Matrix3D (length, width, height);

   this->biasMatrixes = new Matrix3D*[1];
	this->biasMatrixes[0] = nullptr;

   this->weights = (WeightBase**) new BasicWeight*[1];
	this->weights[0] = nullptr;

   this->next = (LayerBase**) new BasicLayer*[1];
	this->next[0] = nullptr;
   this->prev = (LayerBase**) new BasicLayer*[1];
	this->prev[0] = nullptr;

	this->layerMatrixCount = 1;
	this->biasCount = 0;
	this->weightsCount = 0;
	this->nextCount = 0;
	this->prevCount = 0;

	this->type = LayerBase::LayerType::Basic;
	this->activationType = activationType;
}


BasicLayer::BasicLayer () {
	this->layerMatrixes = new Matrix3D* [1];
   this->layerMatrixes[0] = nullptr;
	
   this->biasMatrixes = new Matrix3D*[1];
	this->biasMatrixes[0] = nullptr;

   this->weights = (WeightBase**) new BasicWeight*[1];
	this->weights[0] = nullptr;

   this->next = (LayerBase**) new BasicLayer*[1];
	this->next[0] = nullptr;
   this->prev = (LayerBase**) new BasicLayer*[1];
	this->prev[0] = nullptr;

	this->layerMatrixCount = 0;
	this->biasCount = 0;
	this->weightsCount = 0;
	this->nextCount = 0;
	this->prevCount = 0;

	this->type = LayerBase::LayerType::Basic;
	this->activationType = ActivationType::Sigmoid;
}

BasicLayer::~BasicLayer () { 
   if (this->getLayerMatrix() != nullptr) {
		for (int i = 0; i < this->layerMatrixCount; i++) {
			delete this->getLayerMatrix(i);
		}
   }
   if (this->getBias() != nullptr) {
		for (int i = 0; i < this->biasCount; i++) {
			delete this->getBias(i);
		}
   }
   if (this->getWeights() != nullptr) {
		for (int i = 0; i < this->weightsCount; i++) {
			delete this->getWeights(i);
		}
   }
   if (this->getNext() != nullptr) {
		for (int i = 0; i < this->nextCount; i++) {
			delete this->getNext(i);
		}
	}
}

// broken function
BasicLayer::BasicLayer (const BasicLayer& b, bool copyNext) {
	this->layerMatrixCount = b.getLayerMatrixCount();
	this->biasCount = b.getBiasCount();
	this->weightsCount = b.getWeightsCount();
	this->nextCount = b.getNextCount();
	this->prevCount = b.getPrevCount();

	this->type = b.getLayerType();
	this->activationType = b.getActivationType();

	this->layerMatrixes = new Matrix3D* [1];
   this->layerMatrixes[0] = nullptr;
	if (b.getLayerMatrix() == nullptr) {
		this->layerMatrixes[0] = nullptr;
	} else {
		this->setLayerMatrix(new Matrix3D(*b.getLayerMatrix()));
	}

	this->biasMatrixes = new Matrix3D*[1];
	this->biasMatrixes[0] = nullptr;
	if (b.getBias() != nullptr) {
		this->setBias(new Matrix3D(*(b.getBias())));
	}

	this->weights = (WeightBase**) new BasicWeight*[1];
	this->weights[0] = nullptr;
	if (b.getWeights() != nullptr) {
		this->setWeights(new BasicWeight(*(b.getWeights())));
	}

	this->next = (LayerBase**) new BasicLayer*[1];
	this->next[0] = nullptr;
	this->prev = (LayerBase**) new BasicLayer*[1];
	this->prev[0] = nullptr;

	if (copyNext) {
		const BasicLayer* bCurrent = &b;
		BasicLayer* thisCurrent = this;
		while (bCurrent->getNext() != nullptr) {
			bCurrent = bCurrent->getNext();
			thisCurrent->next = (LayerBase**) new BasicLayer*[1];
			thisCurrent->next[0] = nullptr;
			thisCurrent->setNext(new BasicLayer());
			if (bCurrent->getLayerMatrix() != nullptr) {
				thisCurrent->getNext()->setLayerMatrix(new Matrix3D (*bCurrent->getLayerMatrix()));
			}
			if (bCurrent->getBias() != nullptr) {
				thisCurrent->getNext()->setBias(new Matrix3D (*bCurrent->getBias()));
			}
			if (bCurrent->getWeights() != nullptr) {
				thisCurrent->getNext()->setWeights(new BasicWeight (*bCurrent->getWeights()));
			}
			thisCurrent->getNext()->setPrev(thisCurrent);
			thisCurrent = thisCurrent->getNext();
		}
	}
} 

BasicLayer* BasicLayer::getNext (int index) const {
	return (BasicLayer*) this->LayerBase::getNext(index);
}

BasicWeight* BasicLayer::getWeights (int index) const {
	return (BasicWeight*) this->LayerBase::getWeights(index);
}

// BasicLayer* BasicLayer::add (LayerBase* layer) {
//    if (this->getNext() == nullptr) {
//       this->next[0] = layer;
// 		this->biasMatrixes[0] = this->newBias();
// 		this->weights[0] = this->newWeight();
//    } else {
//       this->next[0] = this->getNext()->add(layer);
//    }
//    return this;
// }

// BasicLayer* BasicLayer::add (Matrix3D* layerMatrix, Matrix3D* biasMatrix, BasicWeight* weights) {
//    if (this->getNext() == nullptr) {
//       this->next[0] = new BasicLayer (layerMatrix, nullptr, nullptr);
//       this->getNext()->setPrev(this);
//       if (this->getBias() == nullptr) {
//          this->biasMatrixes[0] = new Matrix3D(this->getNext()->getLayerMatrix()->getLength(), this->getNext()->getLayerMatrix()->getWidth(), this->getNext()->getLayerMatrix()->getHeight());
//          this->getBias()->randomize(-0.05, 0.05);
//       } else {
//          this->getBias()->setMatrix(biasMatrix);
//       }
		
// 		this->weights[0] = this->newWeight();
//       return this;
//    }
//    this->getNext()->add(layerMatrix, biasMatrix, weights);
//    return this;
// }

WeightBase* BasicLayer::newWeight(int index) {
	if (this->getNext() == nullptr) {
		return nullptr;
	}

	return new BasicWeight (
		this->getLayerMatrix(0)->getLength(), 
		this->getLayerMatrix(0)->getWidth(),
		this->getLayerMatrix(0)->getHeight(),
		this->getNext(index)->getLayerMatrix(0)->getLength(),
		this->getNext(index)->getLayerMatrix(0)->getWidth(),
		this->getNext(index)->getLayerMatrix(0)->getHeight(),
		1);
}

Matrix3D* BasicLayer::newBias(int index) {
	if (this->getNext() == nullptr) {
		return nullptr;
	}

	return new Matrix3D (
		this->getNext()->getLayerMatrix(index)->getLength(),
		this->getNext()->getLayerMatrix(index)->getWidth(),
		this->getNext()->getLayerMatrix(index)->getHeight()
	);
}

void artificialIntelligence::classes::BasicLayer::calculateAndUpdateAllCPU () {
   if (this->getNext(0) == nullptr) {
      return;
   }
   this->calculateAndUpdateLayerCPU();
   this->getNext(0)->calculateAndUpdateAllCPU();
}

void BasicLayer::calculateAndUpdateLayerCPU () {
   Matrix3D* nextLayer = this->getNext(0)->getLayerMatrix(0);
   Matrix3D* outputs = new Matrix3D (nextLayer->getLength(), nextLayer->getWidth(), nextLayer->getHeight());
	outputs->setAll(0);
   if (isnan(*outputs->getData(0, 0, 0))) {
      std::cout << "null init";
      exit (0);
   }
	// nextLayer->printMatrix();

   float activation = 0;
   for (int fl = 0; fl < this->getLayerMatrix(0)->getLength(); fl++) {
      for (int fw = 0; fw < this->getLayerMatrix(0)->getWidth(); fw++) {
         for (int fh = 0; fh < this->getLayerMatrix(0)->getHeight(); fh++) {
            for (int sl = 0; sl < nextLayer->getLength(); sl++) {
               for (int sw = 0; sw < nextLayer->getWidth(); sw++) {
                  for (int sh = 0; sh < nextLayer->getHeight(); sh++) {
                     activation = *this->getLayerMatrix(0)->getData(fl, fw, fh) * *this->getWeights(0)->getData(fl, fw, fh, sl, sw, sh) + *outputs->getData(sl, sw, sh);
                     outputs->insert(activation, sl, sw, sh);
                  }
               }
            }
         }
      }
   } 

	// std::cout << "acc: " << this->activationType << '\n';
	// this->getNext(0)->setLayerMatrix(*this->getNext(0)->getLayerMatrix() + this->getBias(0));
	// activate (this->activationType, this->getNext(0)->getLayerMatrix(0));
	*outputs += this->getBias(0);
	activate(this->activationType, outputs);
   // for (int sl = 0; sl < nextLayer->getLength(); sl++) {
   //    for (int sw = 0; sw < nextLayer->getWidth(); sw++) {
   //       for (int sh = 0; sh < nextLayer->getHeight(); sh++) {
   //          activation = activate(this->activationType, *outputs->getData(sl, sw, sh) + *this->getBias(0)->getData(sl, sw, sh));
   //          outputs->insert(activation, sl, sw, sh);
   //       }
   //    }
   // }

   this->getNext(0)->setLayerMatrix (outputs);
}

void BasicLayer::calculateAndUpdateAllGPUV2() {
	BasicLayer* currentLayer = this;
	Matrix3D* currentLayerMatrix = currentLayer->getLayerMatrix();

	long long numInputs = currentLayerMatrix->getSize() / sizeof(float);
	long long numOutputs = currentLayer->getNext()->getLayerMatrix()->getSize() / sizeof(float);
	long long numWeights = numInputs * numOutputs;
	long long numOutputsRemaining = numOutputs;
	long long outputIndex = 0;

	long long numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs; 
	long long numThreads = 512;
	long long maxWeightIndex = currentLayer->getWeights()->getWeightMatrix()->getSize() / sizeof(float);
	long long numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
	long long sharedSize = numThreads * sizeof(float); 
	if (maxWeightIndex > numWeights) {
		maxWeightIndex = numWeights;
	}

	float* input = currentLayerMatrix->getArr();
	float* output = currentLayer->getNext()->getLayerMatrix()->getArr();
	float* current_input;
	float* current_output;
	gpuErrchk(hipMalloc((void **) &current_input, currentLayerMatrix->getSize()));
	gpuErrchk(hipMalloc((void **) &current_output, numOutputs * sizeof(float)));

	// streams for asynchronous
	hipStream_t stream1, stream2;
	hipStreamCreate ( &stream1); 
	hipStreamCreate ( &stream2); 
	
	BasicWeight* currentWeight = currentLayer->getWeights();
	long long currentWeightMatrixIndex = 0;
	long long weightsAddedLastSet = 0;
	long long weightsInCurrentKernelRun = 0;

	float* current_weights;
	float* next_weights;

	gpuErrchk(hipMalloc((void **) &current_weights, maxWeightIndex * sizeof(float)));
	gpuErrchk(hipMalloc((void **) &next_weights, maxWeightIndex * sizeof(float)));
	gpuErrchk(hipMemcpy(current_weights, currentWeight->getWeightMatrix(0)->getArr(), maxWeightIndex * sizeof(float), hipMemcpyHostToDevice));
	weightsInCurrentKernelRun = maxWeightIndex;
	weightsAddedLastSet = maxWeightIndex;
	currentWeightMatrixIndex++;
	
	int startingOutputID = 0;
	int nextOutputID = maxWeightIndex % currentWeight->getOutputSize();
	int numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;

	gpuErrchk(hipMemcpy(current_input, input, currentLayerMatrix->getSize(), hipMemcpyHostToDevice)); 
	gpuErrchk(hipMemset(current_output, 0b00000000, numOutputs * sizeof(float)));

	int debugCounter = 0;

	numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;
   while (currentLayer->getNext() != nullptr) {
		currentWeightMatrixIndex = 1;
		outputIndex = 0;
		startingOutputID = 0;
		numOutputsRemaining = numOutputs;
		nextOutputID = weightsAddedLastSet;
		
		bool weightsFinished = false;
		long long weightsUsed = 0;
		do {
			
			if (numWeightsMatrixesLeft >= 1){
				if (currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float) < maxWeightIndex) {
					maxWeightIndex = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
				}
				gpuErrchk(hipMemcpyAsync(next_weights, currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr(), maxWeightIndex * sizeof(float), hipMemcpyHostToDevice));
				weightsAddedLastSet = maxWeightIndex;
				currentWeightMatrixIndex++;
				numWeightsMatrixesLeft -= 1;
			} 
			
			else { 
				if (currentLayer->getNext()->getNext() != nullptr) {
					int nextNumWeights = numOutputs * currentLayer->getNext()->getNext()->getLayerMatrix()->getSize() / sizeof(float);
					int nextMaxWeightIndex = currentLayer->getNext()->getWeights()->getWeightMatrix(0)->getSize() / sizeof(float);
					if (nextMaxWeightIndex > nextNumWeights) {
						nextMaxWeightIndex = nextNumWeights;
					}
					
					gpuErrchk(hipFree(next_weights));
					gpuErrchk(hipMalloc((void **) &next_weights, nextMaxWeightIndex * sizeof(float)));
					gpuErrchk(hipMemcpyAsync(next_weights, currentLayer->getNext()->getWeights()->getWeightMatrix(0)->getArr(), nextMaxWeightIndex * sizeof(float), hipMemcpyHostToDevice));
					currentWeightMatrixIndex = 1;
					numWeightsMatrixesLeft = std::ceil((float)nextNumWeights / nextMaxWeightIndex) - 1;
					weightsAddedLastSet = nextMaxWeightIndex;
				}
				weightsFinished = true;
			}
			
			long long helper = 0;

			do {
				if (numOutputsRemaining > 0) {
					// std::cout << "inside22\n";
					// std::cout << "numBlocks: " << numBlocks << '\n';
					// std::cout << "numOutputs: " << numOutputs << '\n';
					// std::cout << "numPerThread: " << numPerThread << '\n';
					// std::cout << "weightsInCurrentKernelRun: " << weightsInCurrentKernelRun << "\n";
					// std::cout << "numOutputsRemaining: " << numOutputsRemaining << '\n';
					// std::cout << "helper: " << helper << '\n';
					// std::cout << "weightsUsed: " << weightsUsed << "\n";
					// std::cout << "numWeightsMatrixesLeft: " << numWeightsMatrixesLeft << "\n";
					// std::cout << "weightsAddedLastSet: " << weightsAddedLastSet << "\n";
					// std::cout << "startingOutputID: " << startingOutputID << "\n\n";
					
					if (numOutputsRemaining - numBlocks < 0) {
						numBlocks = numOutputsRemaining;
					}

					artificialIntelligence::classes::calculateAndUpdateLayerGPUBasic<<< numBlocks, numThreads, sharedSize, stream1 >>>(current_input, current_weights, current_output, numBlocks, numOutputs, numPerThread, weightsInCurrentKernelRun, helper, weightsUsed, startingOutputID);
					outputIndex += numBlocks;
					numOutputsRemaining -= numBlocks;
				}
				startingOutputID += numBlocks;
				helper += numBlocks;

			} while (numOutputsRemaining > 0);
			gpuErrchk(hipDeviceSynchronize());
			
			startingOutputID = nextOutputID % numOutputs;
			nextOutputID += weightsInCurrentKernelRun % numOutputs;
			numOutputsRemaining = numOutputs;

			weightsUsed += weightsInCurrentKernelRun;
			numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs;
			weightsInCurrentKernelRun = weightsAddedLastSet;

			float* temp = current_weights;
			current_weights = next_weights;
			next_weights = temp;

		} while (!weightsFinished);
		
		gpuErrchk(hipMemcpy(output, current_output, numOutputs * sizeof(float), hipMemcpyDeviceToHost));
		// printArr(currentWeight->getWeightMatrix(currentWeightMatrixIndex - 1)->getArr(), 10);
		Matrix3D* bias = currentLayer->getBias();
		currentLayer = currentLayer->getNext();
		currentLayerMatrix = currentLayer->getLayerMatrix();
		currentWeight = currentLayer->getWeights();
		numInputs = currentLayerMatrix->getSize() / sizeof(float);

		if (currentLayer->getNext() != nullptr) {
			output = currentLayer->getNext()->getLayerMatrix()->getArr();
			numOutputs = currentLayer->getNext()->getLayerMatrix()->getSize() / sizeof(float);
			numWeights = numInputs * numOutputs;
			maxWeightIndex = currentLayer->getWeights()->getWeightMatrix()->getSize();
			numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs;
			numThreads = 512; // arbitrary
			numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
			output = currentLayer->getNext()->getLayerMatrix()->getArr();
			gpuErrchk(hipFree(next_weights));
			gpuErrchk(hipMalloc((void **) &next_weights, maxWeightIndex * sizeof(float)));
			gpuErrchk(hipFree(current_output));
			gpuErrchk(hipMalloc((void **) &current_output, numOutputs * sizeof(float)));
			gpuErrchk(hipMemset(current_output, 0b00000000, numOutputs * sizeof(float))); 
		}
		
		*currentLayer->getLayerMatrix() += bias;
		// std::cout << "acc: " << currentLayer->getPrev(0)->getActivationType() << '\n';
		activate (currentLayer->getPrev(0)->getActivationType(), currentLayer->getLayerMatrix(0));
		gpuErrchk(hipFree(current_input));
		gpuErrchk(hipMalloc((void **) &current_input, currentLayerMatrix->getSize()));
		input = currentLayerMatrix->getArr();
		gpuErrchk(hipMemcpy(current_input, input, currentLayerMatrix->getSize(), hipMemcpyHostToDevice));

		debugCounter++;
	}
	gpuErrchk(hipFree(current_input));	
	gpuErrchk(hipFree(current_output));
	gpuErrchk(hipFree(current_weights));	
	gpuErrchk(hipFree(next_weights));
	gpuErrchk(hipStreamDestroy(stream1));
	gpuErrchk(hipStreamDestroy(stream2));
}

__global__ void artificialIntelligence::classes::calculateAndUpdateLayerGPUBasic(float* nodeValues, float* weights, float* output, int inputSize, int outputSize, int numPerThread, long long maxWeightIndex, long long helperIndex, long long startingWeight, int startingOutputId) {
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned long long outputNodeId = (blockIdx.x + startingOutputId) % outputSize;
	unsigned int numThreads = blockDim.x;
	unsigned long long weightIndex = tid * outputSize + blockIdx.x + helperIndex;
	unsigned long long inputNodeId = 0;
	unsigned int gridSize = numThreads*outputSize;
	sdata[tid] = 0;

	while (weightIndex < maxWeightIndex) {
		inputNodeId = (weightIndex + startingWeight) / outputSize;
		sdata[tid] += nodeValues[inputNodeId] * weights[weightIndex];
		weightIndex += gridSize;
	}

	__syncthreads();

	for (unsigned int s=numThreads/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	
	if (tid == 0) {
		output[outputNodeId] += sdata[0];
	}
}

Matrix3D* BasicLayer::calculateErrorCPU (Matrix3D* delta) {
	Matrix3D* currentLayerMatrix = this->getLayerMatrix();
	Matrix3D* error = new Matrix3D(currentLayerMatrix->getLength(), currentLayerMatrix->getWidth(), currentLayerMatrix->getHeight());
	for (int l = 0; l < currentLayerMatrix->getLength(); l++) {
		for (int w = 0; w < currentLayerMatrix->getWidth(); w++) {
			for (int h = 0; h < currentLayerMatrix->getHeight(); h++) {
				Matrix3D* outputMatrix = this->getNext(0)->getLayerMatrix(0);
				Matrix3D* weightedMatrix = new Matrix3D (delta->getLength(), delta->getWidth(), delta->getHeight());
				for (int l2 = 0; l2 < outputMatrix->getLength(); l2++) {
					for (int w2 = 0; w2 < outputMatrix->getWidth(); w2++) {
						for (int h2 = 0; h2 < outputMatrix->getHeight(); h2++) {
							weightedMatrix->insert(*this->getWeights(0)->getData(l, w, h, l2, w2, h2) * *delta->getData(l2, w2, h2), l2, w2, h2);
						}
					}
				}
				error->insert(weightedMatrix->sum(), l, w, h);
				delete weightedMatrix;
			}
		}
	}
	return error;
}

Matrix3D* BasicLayer::calculateErrorGPU (Matrix3D* delta) {
	BasicLayer* currentLayer = this;
	Matrix3D* currentLayerMatrix = currentLayer->getLayerMatrix();

	long long numInputs = currentLayerMatrix->getSize() / sizeof(float);
	long long numOutputs = currentLayer->getNext()->getLayerMatrix()->getSize() / sizeof(float);
	long long numWeights = numInputs * numOutputs;
	long long numInputsRemaining = numInputs;
	long long inputIndex = 0;
	long long numBlocks = numInputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numInputs; 
	long long numThreads = 512;
	long long maxWeightIndex = numBlocks * numOutputs;
	long long numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
	long long sharedSize = numThreads * sizeof(float); 
	if (maxWeightIndex > numWeights) {
		maxWeightIndex = numWeights;
	}
	Matrix3D* errorMatrix = new Matrix3D(currentLayer->getLayerMatrix()->getLength(), currentLayer->getLayerMatrix()->getWidth(), currentLayer->getLayerMatrix()->getHeight());
	float* error = errorMatrix->getArr();
	float* current_error;
	float* current_delta;
	gpuErrchk(hipMalloc((void **) &current_error, errorMatrix->getSize()));
	gpuErrchk(hipMalloc((void **) &current_delta, delta->getSize()));
	gpuErrchk(hipMemcpy(current_error, error, errorMatrix->getSize(), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(current_delta, delta->getArr(), delta->getSize(), hipMemcpyHostToDevice));

	hipStream_t stream1, stream2;
	hipStreamCreate ( &stream1); 
	hipStreamCreate ( &stream2); 
	
	BasicWeight* currentWeight = currentLayer->getWeights();
	long long matrixSize = currentWeight->getWeightMatrix()->getSize() / sizeof(float);
	long long currentWeightMatrixIndex = 0;
	long long weightsAddedLastSet = 0;
	long long weightsInCurrentKernelRun = 0;

	// std::cout << "Number of threads: " << numThreads << '\n';
	// std::cout << "Number of blocks: " << numBlocks << '\n';
	// std::cout << "Number per thread: " << numPerThread << '\n';
	// std::cout << "Number of bytes for shared storage: " << sharedSize << "\n";
	// std::cout << "Max array index: " << maxWeightIndex << "\n";
	// std::cout << "Max byte index: " << maxWeightIndex * sizeof(float) << "\n";
	// std::cout << "numLeftToAdd: " <<  numLeftToAdd << "\n";

	float* current_weights;
	float* next_weights;

	gpuErrchk(hipMalloc((void **) &current_weights, maxWeightIndex * sizeof(float)));
	gpuErrchk(hipMalloc((void **) &next_weights, maxWeightIndex * sizeof(float)));

	int weightsInCurrentMatrix = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
	int weightsInBasicWeight = currentWeight->getSize();

	int numberOfWeightsToAdd = numBlocks * numOutputs;
	int toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;

	int amountAdded = 0;
	int weightsAdded = 0;

	while (numberOfWeightsToAdd > 0) {
		toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;
		// std::cout << "\ntoAdd: " <<  toAdd << "\n";
		// std::cout << "numberOfWeightsToAdd: " <<  numberOfWeightsToAdd << "\n";
		// std::cout << "weightsInCurrentMatrix: " <<  weightsInCurrentMatrix << "\n";
		// std::cout << "currentWeightMatrixIndex: " <<  currentWeightMatrixIndex << "\n";
		gpuErrchk(hipMemcpy(&current_weights[weightsAdded], &currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded], toAdd * sizeof(float), hipMemcpyHostToDevice));
		if (toAdd == weightsInCurrentMatrix) {
			currentWeightMatrixIndex++;
			// std::cout << "inside1\n";
			numberOfWeightsToAdd -= toAdd;
			amountAdded = 0;
			weightsAdded += toAdd;
			if (weightsAdded < numWeights) {
				weightsInCurrentMatrix = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
			}
		} else {
			amountAdded = toAdd;
			numberOfWeightsToAdd = 0;
			weightsInCurrentMatrix -= toAdd;
			weightsAdded += toAdd;
		}
	}
	weightsInCurrentKernelRun = weightsAdded;
	weightsAddedLastSet = weightsAdded;
	
	int startingInputID = 0;
	int numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;

	numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;
	inputIndex = 0;
	startingInputID = 0;
	numInputsRemaining = numInputs;
	bool weightsFinished = false;
	long long weightsUsed = 0;
	do {
		if (numInputsRemaining > 0) {
			numBlocks = (weightsUsed + weightsInCurrentKernelRun) * numInputs / numWeights - weightsUsed * numInputs / numWeights;
			// std::cout << "inside22\n";
			// std::cout << "numBlocks: " << numBlocks << '\n';
			// std::cout << "numInputs: " << numInputs << '\n';
			// std::cout << "numPerThread: " << numPerThread << '\n';
			// std::cout << "weightsInCurrentKernelRun: " << weightsInCurrentKernelRun << "\n";
			// std::cout << "numInputsRemaining: " << numInputsRemaining << '\n';
			// std::cout << "weightsUsed: " << weightsUsed << "\n";
			// std::cout << "startingInputID: " << startingInputID << "\n\n";

			if (numInputsRemaining - numBlocks < 0) {
				numBlocks = numInputsRemaining;
			}
			
			artificialIntelligence::classes::calculateErrorBasic<<< numBlocks, numThreads, sharedSize, stream1 >>>(current_weights, current_delta, current_error, numInputs, numOutputs, numPerThread, weightsInCurrentKernelRun, numWeights, weightsUsed, startingInputID);
			inputIndex += numBlocks;
			numInputsRemaining -= numBlocks;
		}
		gpuErrchk(hipDeviceSynchronize());
		startingInputID += numBlocks;

		weightsUsed += weightsInCurrentKernelRun;
		weightsInCurrentKernelRun = weightsAddedLastSet;
		if (numWeights - weightsAdded > 0) {
			// std::cout << "here\n";
			// exit(0);
			// asynchronously add the next set of weights
			numBlocks = numInputsRemaining > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numInputsRemaining;
			numberOfWeightsToAdd = numBlocks * numOutputs;
			toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;
			amountAdded = weightsAdded % (WEIGHT_MAX_SIZE);
			int weightCounter = 0;
			if (weightsInCurrentMatrix > 0) {
				weightsInCurrentKernelRun = numberOfWeightsToAdd;
				while (numberOfWeightsToAdd > 0) {
					toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;
					// std::cout << "currentWeightMatrixIndex: " << currentWeightMatrixIndex << "\n";
					// std::cout << "amountAdded: " << amountAdded << "\n";
					// std::cout << "toAdd: " << toAdd << "\n";
					// std::cout << "weightsInCurrentMatrix: " << weightsInCurrentMatrix << "\n";
					// std::cout << "currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded]: " << currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded] << '\n';
					gpuErrchk(hipMemcpyAsync(&next_weights[weightCounter], &currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded], toAdd * sizeof(float), hipMemcpyHostToDevice));
					if (toAdd == weightsInCurrentMatrix) {
						currentWeightMatrixIndex++;
						numberOfWeightsToAdd -= toAdd;
						amountAdded = 0;
						weightsAdded += toAdd;
						weightCounter += toAdd;
						if (weightsAdded < numWeights) {
							weightsInCurrentMatrix = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
						}
					} else {
						numberOfWeightsToAdd = 0;
						weightsInCurrentMatrix -= toAdd;
						weightsAdded += toAdd;
					}
				}
			}
		}
		else {
			weightsFinished = true;
		}

		gpuErrchk(hipDeviceSynchronize());

		float* temp = current_weights;
		current_weights = next_weights;
		next_weights = temp;

	} while (!weightsFinished);

	gpuErrchk(hipMemcpy(error, current_error, numInputs * sizeof(float), hipMemcpyDeviceToHost));

	// :::: FREE ALL ALLOCATED MEMORY :::: //
	gpuErrchk(hipFree(current_error));	
	gpuErrchk(hipFree(current_delta));
	gpuErrchk(hipFree(current_weights));	
	gpuErrchk(hipFree(next_weights));
	gpuErrchk(hipStreamDestroy(stream1));
	gpuErrchk(hipStreamDestroy(stream2));
	return errorMatrix;
}

__global__ void artificialIntelligence::classes::calculateErrorBasic(float* weights, float* delta, float* error, int inputSize, int outputSize, int numPerThread, long long maxWeightIndex, long long helperIndex, long long startingWeight, int startingInputID) {
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int numThreads = blockDim.x;
	unsigned long long inputNodeId = blockIdx.x + startingInputID;
	unsigned long long weightIndex = tid + blockIdx.x * outputSize;
	unsigned int gridSize = numThreads;
	int weightsToAddStart = outputSize * (blockIdx.x);
	int weightsToAddEnd = outputSize * (blockIdx.x + 1);

	sdata[tid] = 0;
	while (weightIndex >= weightsToAddStart && weightIndex < weightsToAddEnd) {
		sdata[tid] += weights[weightIndex] * delta[(startingWeight + weightIndex) % outputSize];
		weightIndex += gridSize;
	}

	__syncthreads();

	for (unsigned int s=numThreads/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	
	if (tid == 0) {
		error[inputNodeId] += sdata[0];
	}
}

void BasicLayer::updateWeightsCPU (Matrix3D* delta, double learningRate) {
	Matrix3D* currentLayerMatrix = this->getLayerMatrix();
	for (int l = 0; l < currentLayerMatrix->getLength(); l++) {
		for (int w = 0; w < currentLayerMatrix->getWidth(); w++) {
			for (int h = 0; h < currentLayerMatrix->getHeight(); h++) {
				float inputValue = *currentLayerMatrix->getData(l, w, h);
				float value = 0;
				
				Matrix3D* weightMatrix = this->getNext()->getLayerMatrix();
				for (int l2 = 0; l2 < weightMatrix->getLength(); l2++) {
					for (int w2 = 0; w2 < weightMatrix->getWidth(); w2++) {
						for (int h2 = 0; h2 < weightMatrix->getHeight(); h2++) {
							value = *this->getWeights()->getData(l, w, h, l2, w2, h2) + inputValue * *delta->getData(l2, w2, h2) * learningRate;
							this->getWeights()->insertData(value, l, w, h, l2, w2, h2);
						}
					}
				}
			}
		}
	}
}

void BasicLayer::updateWeightsGPU (Matrix3D* delta, double learningRate) {
	BasicLayer* currentLayer = this;
	Matrix3D* currentLayerMatrix = currentLayer->getLayerMatrix();

	long long numInputs = currentLayerMatrix->getSize() / sizeof(float);
	long long numOutputs = currentLayer->getNext()->getLayerMatrix()->getSize() / sizeof(float);
	long long numWeights = numInputs * numOutputs;
	long long inputIndex = 0;
	long long numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs; 
	long long numThreads = 512;
	long long maxWeightIndex = numBlocks * numOutputs;
	long long numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
	long long sharedSize = numThreads * sizeof(float); 
	if (maxWeightIndex > numWeights) {
		maxWeightIndex = numWeights;
	}
	
	hipStream_t stream1, stream2;
	hipStreamCreate ( &stream1); 
	hipStreamCreate ( &stream2);

	BasicWeight* currentWeight = currentLayer->getWeights();
	long long matrixSize = currentWeight->getWeightMatrix()->getSize() / sizeof(float);
	long long currentWeightMatrixIndex = 0;
	long long weightsInCurrentKernelRun = 0;
	
	int weightsInCurrentMatrix = currentWeight->getWeightMatrix(0)->getSize() / sizeof(float);

	Matrix3D* inputMatrix = currentLayer->getLayerMatrix();
	float* current_input;
	float* current_delta;
	gpuErrchk(hipMalloc((void **) &current_input, inputMatrix->getSize()));
	gpuErrchk(hipMalloc((void **) &current_delta, delta->getSize()));
	gpuErrchk(hipMemcpy(current_input, inputMatrix->getArr(), inputMatrix->getSize(), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(current_delta, delta->getArr(), delta->getSize(), hipMemcpyHostToDevice));
	

	float* current_weights;
	float* next_weights;
	gpuErrchk(hipMalloc((void **) &current_weights, currentWeight->getWeightMatrix(0)->getSize()));
	gpuErrchk(hipMalloc((void **) &next_weights, currentWeight->getWeightMatrix(0)->getSize()));
	gpuErrchk(hipMemcpy(current_weights, currentWeight->getWeightMatrix(0)->getArr(), currentWeight->getWeightMatrix(0)->getSize(), hipMemcpyHostToDevice));
	weightsInCurrentKernelRun = currentWeight->getWeightMatrix(0)->getSize() / sizeof(float);

	long long weightsUsed = 0;
	int startingInputId = 0;
	while ((numWeights - weightsUsed) != 0) {
		// std::cout << "inside22\n";
		// std::cout << "numBlocks: " << numBlocks << '\n';
		// std::cout << "numOutputs: " << numOutputs << '\n';
		// std::cout << "numPerThread: " << numPerThread << '\n';
		// std::cout << "weightsInCurrentKernelRun: " << weightsInCurrentKernelRun << "\n";
		// std::cout << "numOutputsRemaining: " << numOutputsRemaining << '\n';
		// std::cout << "weightsUsed: " << weightsUsed << "\n";
		// std::cout << "startingInputId: " << startingInputId << "\n\n";

		artificialIntelligence::classes::updateWeightsBasic<<<numBlocks, numThreads, sharedSize, stream1>>>(current_weights, current_delta, current_input, numInputs, numOutputs, numPerThread, weightsInCurrentKernelRun, numWeights, weightsUsed, startingInputId, learningRate);
		inputIndex += numBlocks;
		
		startingInputId = weightsUsed / numOutputs;

		weightsUsed += currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);

		currentWeightMatrixIndex++;
		if ((numWeights - weightsUsed) != 0) {
			gpuErrchk(hipMemcpyAsync(next_weights, currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr(), currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize(), hipMemcpyHostToDevice));
			weightsInCurrentKernelRun = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
		}

		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(currentWeight->getWeightMatrix(currentWeightMatrixIndex - 1)->getArr(), current_weights, currentWeight->getWeightMatrix(currentWeightMatrixIndex - 1)->getSize(), hipMemcpyDeviceToHost));

		float* temp = current_weights;
		current_weights = next_weights;
		next_weights = temp;
	}

	gpuErrchk(hipFree(current_input));
	gpuErrchk(hipFree(current_delta));
	gpuErrchk(hipFree(current_weights));	
	gpuErrchk(hipFree(next_weights));
	gpuErrchk(hipStreamDestroy(stream1));
	gpuErrchk(hipStreamDestroy(stream2));
}

__global__ void artificialIntelligence::classes::updateWeightsBasic(float* weights, float* delta, float* input, int inputSize, int outputSize, int numPerThread, long long maxWeightIndex, long long helperIndex, long long startingWeight, int startingInputID, double learningRate) {
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int numThreads = blockDim.x;
	unsigned long long weightIndex = tid + numThreads * blockIdx.x;
	unsigned long long outputNodeId = (weightIndex + startingWeight) % outputSize;
	unsigned long long inputNodeId = (weightIndex + startingWeight) / outputSize;
	unsigned int gridSize = numThreads * gridDim.x;
	while (weightIndex < maxWeightIndex) {
		weights[weightIndex] += input[inputNodeId] * delta[outputNodeId] * learningRate;
		weightIndex += gridSize;
		inputNodeId = (weightIndex + startingWeight) / outputSize;
		outputNodeId = (weightIndex + startingWeight) % outputSize;
	}
}

void BasicLayer::printDetails () {
	std::cout << "Basic Fully Connected Layer :: ";
	this->getLayerMatrix()->printMatrixSize();
}  


void BasicLayer::toFile (std::ofstream* outputFile) {
	char* output = new char[sizeof(int) * 6];

	//layer
	*outputFile << this->type << '\n';
   *outputFile << this->getLayerMatrix()->getLength() << ',' << this->getLayerMatrix()->getWidth() << ',' << this->getLayerMatrix()->getHeight() << '\n';

	//bias
   if (this->biasMatrixes[0] == nullptr) {
      return;
   }
   *outputFile << this->getBias()->getLength() << ',' << this->getBias()->getWidth() << ',' << this->getBias()->getHeight() << '\n';
   for (int i = 0; i < this->getBias()->getLength(); i++) {
      for (int j = 0; j < this->getBias()->getWidth(); j++) {
         for (int k = 0; k < this->getBias()->getHeight(); k++) {
            *outputFile << *this->getBias()->getData(i, j, k) << ',';
         }
      }
   }

   outputFile->seekp((int) outputFile->tellp() - 1);
   outputFile->write("\n", 1);

	//weights
   if (this->weights[0] == nullptr) {
      return;
   }
   *outputFile << this->getLayerMatrix()->getLength() << ',' << this->getLayerMatrix()->getWidth() << ',' << this->getLayerMatrix()->getHeight() << ',';
   *outputFile << this->getBias()->getLength() << ',' << this->getBias()->getWidth() << ',' << this->getBias()->getHeight() << '\n';

	int currentWeightMatrix = 0;
	float* weights;
	while (this->getWeights()->getWeightMatrix(currentWeightMatrix) != nullptr) {
		int size = this->getWeights()->getWeightMatrix(currentWeightMatrix)->getSize() + sizeof(float);
		char* output = new char[size];
		char* ptr = output;

		weights = this->getWeights()->getWeightMatrix(currentWeightMatrix)->getArr();
		for (int i = 0, cc = this->getWeights()->getWeightMatrix(currentWeightMatrix)->getSize() / sizeof(float); i < cc; i++) {
			memcpy(ptr, &weights[i], sizeof(float));
			ptr += sizeof(float);
		}
		outputFile->write(output, size);
		currentWeightMatrix++;

		free(output);
	}
	outputFile->seekp((int) outputFile->tellp() - 1);
   outputFile->write("\n", 1);

	//activation
	*outputFile << (int) this->activationType << '\n';

   if (this->getNext() == nullptr) {
      return;
   }
   this->getNext()->toFile(outputFile);
}


LayerBase* BasicLayer::loadFromFile (std::ifstream* inputFile, LayerBase* prev) {
	std::cout << "Loading layer from file\n";
   BasicLayer* layer = new BasicLayer ();
   std::string line;
   getline (*inputFile, line);
	std::cout << "line: " << line << '\n';
   std::stringstream lineStream;
   lineStream << line;
   std::string value;
   getline(lineStream, value, ',');
	std::cout << "v1: " << value << '\n';
   int layerLength = stoi(value);
   getline(lineStream, value, ',');
	std::cout << "v2: " << value << '\n';
   int layerWidth = stoi(value);
   getline(lineStream, value, ',');
	std::cout << "v3: " << value << '\n';
   int layerHeight = stoi(value);
   layer->setLayerMatrix(new Matrix3D (layerLength, layerWidth, layerHeight), 0);
   layer->setPrev(prev, 0);

   lineStream.str(std::string());
   lineStream.clear();
   getline (*inputFile, line);
   lineStream << line;

   if (inputFile->eof()) {
		layer->biasMatrixes = new Matrix3D*[1];
   	layer->weights = (WeightBase**) new BasicWeight*[1];
      return layer;
   }

   getline(lineStream, value, ',');
   int biasLength = stoi(value);
   getline(lineStream, value, ',');
   int biasWidth = stoi(value);
   getline(lineStream, value, ',');
   int biasHeight = stoi(value);
   layer->setBias (new Matrix3D (biasLength, biasWidth, biasHeight));

   lineStream.str(std::string());
   lineStream.clear();
   getline (*inputFile, line);
   lineStream << line;
   for (int i = 0; i < layer->getBias(0)->getLength(); i++) {
      for (int j = 0; j < layer->getBias(0)->getWidth(); j++) {
         for (int k = 0; k < layer->getBias(0)->getHeight(); k++) {
            std::getline(lineStream, value, ',');
            layer->getBias(0)->insert (stod(value), i, j, k);
         }
      }
   }

   getline (*inputFile, line);

   if (inputFile->eof()) {
		layer->weights = (WeightBase**) new BasicWeight*[1];
      return layer;
   }

   BasicWeight* weights = new BasicWeight (
      layer->getLayerMatrix()->getLength(), 
      layer->getLayerMatrix()->getWidth(), 
      layer->getLayerMatrix()->getHeight(), 
      layer->getBias()->getLength(), 
      layer->getBias()->getWidth(), 
      layer->getBias()->getHeight(),
		0
   );

   
   lineStream.str(std::string());
   lineStream.clear();
	
	std::cout << "Inserting weights\n";

	int currentWeightMatrix = 0;
	while (weights->getWeightMatrix(currentWeightMatrix) != nullptr) {
		inputFile->read((char*) weights->getWeightMatrix(currentWeightMatrix)->getArr(), weights->getWeightMatrix(currentWeightMatrix)->getSize());
		currentWeightMatrix++;
	}
	getline(*inputFile, line);

	std::cout << "Finished weights\n";

   layer->setWeights (weights);
	std::cout << line << '\n';
	std::cout << "here\n";
   getline (*inputFile, line);
	std::cout << line << '\n';
	std::cout << "here\n";
   layer->setActivation((ActivationType) stoi(line));

   return layer;
}