#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include <cmath>
#include <unistd.h>

#include <coreutils/classes/matrixes/Matrix3D.cuh>
#include <coreutils/functions/debug/print.hpp>
#include <coreutils/util/time.hpp>
#include <coreutils/util/cudaErrors.cuh>

#include "../../functions/activationFunctions.cuh"
#include "../layers/ConvLayer.cuh"
#include "../weights/ConvWeight.cuh"

using namespace std;
using namespace coreutils::classes::matrixes;
using namespace coreutils::functions::debug;
using namespace artificialIntelligence::classes;
using namespace artificialIntelligence::functions::activation;

#define MAX_BLOCK_SIZE 8192

ConvLayer::ConvLayer (Matrix3D* layerMatrix, Matrix3D* biasMatrix, ConvWeight* weights, ActivationType activationType) {
   // this->layerMatrix = new Matrix3D(layerMatrix->getLength(), layerMatrix->getWidth(), layerMatrix->getHeight());
   // this->getLayerMatrix()->setMatrix(layerMatrix);
	// this->biasMatrixes = new Matrix3D*[1];
	// this->biasMatrixes[0] = nullptr;
   // if (biasMatrix != nullptr) {
   //    this->biasMatrixes[0] = new Matrix3D(biasMatrix->getLength(), biasMatrix->getWidth(), biasMatrix->getHeight());
   //    this->getBias()->setMatrix(biasMatrix);
   // }
	// this->weights = (WeightBase**) new ConvWeight*[1];
   // this->weights[0] = weights;
   // this->next = (LayerBase**) new ConvLayer*[1];
	// this->next[0] = nullptr;
   // this->prev = (LayerBase**) new ConvLayer*[1];
	// this->prev[0] = nullptr;
	// this->type = LayerBase::LayerType::Conv;
}


ConvLayer::ConvLayer (int length, int width, int height, int convLength, int convWidth, int convHeight, int features, int stride, ActivationType activationType) {
	this->layerMatrixes = new Matrix3D* [1];
	this->layerMatrixes[0] = new Matrix3D (length, width, height);
	this->getLayerMatrix(0)->randomize();
	this->biasMatrixes = new Matrix3D*[1];
	this->biasMatrixes[0] = nullptr;
	this->weights = (WeightBase**) new ConvWeight*[1];
	this->weights[0] = nullptr;
	this->next = (LayerBase**) new ConvLayer*[1];
	this->next[0] = nullptr;
	this->prev = (LayerBase**) new ConvLayer*[1];
	this->prev[0] = nullptr;

	this->layerMatrixCount = 1;
	this->biasCount = 0;
	this->weightsCount = 0;
	this->nextCount = 0;
	this->prevCount = 0;

	this->activationType = activationType;
	this->type = LayerBase::LayerType::Conv;

	this->convLength = convLength;
	this->convWidth = convWidth;
	this->convHeight = convHeight;
	this->features = features;
	this->stride = stride;
}


ConvLayer::ConvLayer () {
	this->layerMatrixes = new Matrix3D*[1];
   this->layerMatrixes = nullptr;
   this->biasMatrixes = new Matrix3D*[1];
	this->biasMatrixes[0] = nullptr;
   this->weights = (WeightBase**) new ConvWeight*[1];
	this->weights[0] = nullptr;
   this->next = (LayerBase**) new ConvLayer*[1];
	this->next[0] = nullptr;
   this->prev = (LayerBase**) new ConvLayer*[1];
	this->prev[0] = nullptr;

	this->layerMatrixCount = 0;
	this->biasCount = 0;
	this->weightsCount = 0;
	this->nextCount = 0;
	this->prevCount = 0;

	this->activationType = ActivationType::Sigmoid;
	this->type = LayerBase::LayerType::Conv;

	this->convLength = 0;
	this->convWidth = 0;
	this->convHeight = 0;
	this->features = 0;
	this->stride = 0;
}


ConvLayer::~ConvLayer () { 
   if (this->getLayerMatrix() != nullptr) {
		for (int i = 0; i < this->layerMatrixCount; i++) {
			delete this->getLayerMatrix(i);
		}
   }
   if (this->getBias() != nullptr) {
		for (int i = 0; i < this->biasCount; i++) {
			delete this->getBias(i);
		}
   }
   if (this->getWeights() != nullptr) {
		for (int i = 0; i < this->weightsCount; i++) {
			delete this->getWeights(i);
		}
   }
   if (this->getNext() != nullptr) {
		for (int i = 0; i < this->nextCount; i++) {
			delete this->getNext(i);
		}
	}
}

// ConvLayer* ConvLayer::add (LayerBase* layer) {
//    if (this->getNext() == nullptr) {
//       this->next[0] = layer;
// 		this->biasMatrixes[0] = new Matrix3D(this->getNext()->getLayerMatrix()->getLength(), this->getNext()->getLayerMatrix()->getWidth(), this->getNext()->getLayerMatrix()->getHeight());
// 		this->weights[0] = this->newWeight();
//    } else {
//       this->next[0] = this->getNext()->add(layer);
//    }
//    return this;
// }

// ConvLayer* ConvLayer::add (Matrix3D* layerMatrix, Matrix3D* biasMatrix, ConvWeight* weights, int index) {
//    if (this->getNext() == nullptr) {
//       this->next[0] = new ConvLayer (layerMatrix, nullptr, nullptr);
//       this->getNext()->setPrev(this);
//       if (this->getBias() == nullptr) {
//          this->biasMatrixes[0] = new Matrix3D(this->getNext()->getLayerMatrix()->getLength(), this->getNext()->getLayerMatrix()->getWidth(), this->getNext()->getLayerMatrix()->getHeight());
//          this->getBias()->randomize(-0.05, 0.05);
//       } else {
//          this->getBias()->setMatrix(biasMatrix);
//       }
		
// 		this->weights[0] = weights;
//       return this;
//    }
//    this->getNext()->add(layerMatrix, biasMatrix, weights);
//    return this;
// }

WeightBase* ConvLayer::newWeight(int index) {
	return new ConvWeight (
		this->convLength,
		this->convWidth,
		this->convHeight,
		this->features,
		this->stride,
		1
	);
}

Matrix3D* ConvLayer::newBias(int index) {
	if (this->getNext() == nullptr) {
		return nullptr;
	}

	return new Matrix3D (
		this->getNext()->getLayerMatrix(index)->getLength(),
		this->getNext()->getLayerMatrix(index)->getWidth(),
		this->getNext()->getLayerMatrix(index)->getHeight()
	);
}

void artificialIntelligence::classes::ConvLayer::calculateAndUpdateAllCPU () {
   if (this->getNext() == nullptr) {
      return;
   }
   this->calculateAndUpdateLayerCPU();
   this->getNext()->calculateAndUpdateAllCPU();
}

void ConvLayer::calculateAndUpdateLayerCPU () {


   Matrix3D* nextLayer = this->getNext()->getLayerMatrix();
   Matrix3D* outputs = new Matrix3D (nextLayer->getLength(), nextLayer->getWidth(), nextLayer->getHeight());
	outputs->setAll(0);
   if (isnan(*outputs->getData(0, 0, 0))) {
      std::cout << "bad init\n";
      exit (0);
   }
	
	int numPerFeature = this->getWeights()->paramCount();
   float activation = 0;
	// run through each filter
	for (int f = 0; f < this->features; f++) {
		for (int fl = 0; fl < this->getLayerMatrix()->getLength(); fl++) {
			for (int fw = 0; fw < this->getLayerMatrix()->getWidth(); fw++) {
				for (int fh = 0; fh < this->getLayerMatrix()->getHeight(); fh++) {

					for (int sl = 0; sl < nextLayer->getLength(); sl++) {
						for (int sw = 0; sw < nextLayer->getWidth(); sw++) {
							for (int sh = 0; sh < nextLayer->getHeight(); sh++) {
								// activation = *this->getLayerMatrix()->getData(fl, fw, fh) * *this->getWeights()->getData(fl, fw, fh, sl, sw, sh) + *outputs->getData(sl, sw, sh);
								outputs->insert(activation / numPerFeature, sl, sw, sh);
							}
						}
					}
				}
			}
		} 
	}

   for (int sl = 0; sl < nextLayer->getLength(); sl++) {
      for (int sw = 0; sw < nextLayer->getWidth(); sw++) {
         for (int sh = 0; sh < nextLayer->getHeight(); sh++) {
            activation = leakyRelu(*outputs->getData(sl, sw, sh) + *this->getBias()->getData(sl, sw, sh));
            outputs->insert(activation, sl, sw, sh);
         }
      }
   }

   this->getNext()->setLayerMatrix (outputs);
   delete outputs;


}

void ConvLayer::calculateAndUpdateAllGPUV2() {
	// ConvLayer* currentLayer = this;
	// Matrix3D* currentLayerMatrix = currentLayer->getLayerMatrix();

	// long long numInputs = currentLayerMatrix->getSize() / sizeof(float);
	// long long numOutputs = currentLayer->getNext()->getLayerMatrix()->getSize() / sizeof(float);
	// long long numWeights = numInputs * numOutputs;
	// long long numOutputsRemaining = numOutputs;
	// long long outputIndex = 0;

	// long long numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs; 
	// long long numThreads = 512;
	// long long maxWeightIndex = currentLayer->getWeights()->getWeightMatrix()->getSize() / sizeof(float);
	// long long numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
	// long long sharedSize = numThreads * sizeof(float); 
	// if (maxWeightIndex > numWeights) {
	// 	maxWeightIndex = numWeights;
	// }

	// float* input = currentLayerMatrix->getArr();
	// float* output = currentLayer->getNext()->getLayerMatrix()->getArr();
	// float* current_input;
	// float* current_output;
	// gpuErrchk(hipMalloc((void **) &current_input, currentLayerMatrix->getSize()));
	// gpuErrchk(hipMalloc((void **) &current_output, numOutputs * sizeof(float)));

	// // streams for asynchronous
	// hipStream_t stream1, stream2;
	// hipStreamCreate ( &stream1); 
	// hipStreamCreate ( &stream2); 
	
	// ConvWeight* currentWeight = currentLayer->getWeights();
	// long long currentWeightMatrixIndex = 0;
	// long long weightsAddedLastSet = 0;
	// long long weightsInCurrentKernelRun = 0;

	// float* current_weights;
	// float* next_weights;

	// gpuErrchk(hipMalloc((void **) &current_weights, maxWeightIndex * sizeof(float)));
	// gpuErrchk(hipMalloc((void **) &next_weights, maxWeightIndex * sizeof(float)));
	// gpuErrchk(hipMemcpy(current_weights, currentWeight->getWeightMatrix(0)->getArr(), maxWeightIndex * sizeof(float), hipMemcpyHostToDevice));
	// weightsInCurrentKernelRun = maxWeightIndex;
	// weightsAddedLastSet = maxWeightIndex;
	// currentWeightMatrixIndex++;
	
	// int startingOutputID = 0;
	// int nextOutputID = maxWeightIndex % currentWeight->getOutputSize();
	// int numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;

	// gpuErrchk(hipMemcpy(current_input, input, currentLayerMatrix->getSize(), hipMemcpyHostToDevice)); 
	// gpuErrchk(hipMemset(current_output, 0b00000000, numOutputs * sizeof(float)));

	// int debugCounter = 0;

	// numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;
   // while (currentLayer->getNext() != nullptr) {
	// 	currentWeightMatrixIndex = 1;
	// 	outputIndex = 0;
	// 	startingOutputID = 0;
	// 	numOutputsRemaining = numOutputs;
	// 	nextOutputID = weightsAddedLastSet;
		
	// 	bool weightsFinished = false;
	// 	long long weightsUsed = 0;
	// 	do {
			
	// 		if (numWeightsMatrixesLeft >= 1){
	// 			if (currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float) < maxWeightIndex) {
	// 				maxWeightIndex = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
	// 			}
	// 			gpuErrchk(hipMemcpyAsync(next_weights, currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr(), maxWeightIndex * sizeof(float), hipMemcpyHostToDevice));
	// 			weightsAddedLastSet = maxWeightIndex;
	// 			currentWeightMatrixIndex++;
	// 			numWeightsMatrixesLeft -= 1;
	// 		} 
			
	// 		else { 
	// 			if (currentLayer->getNext()->getNext() != nullptr) {
	// 				int nextNumWeights = numOutputs * currentLayer->getNext()->getNext()->getLayerMatrix()->getSize() / sizeof(float);
	// 				int nextMaxWeightIndex = currentLayer->getNext()->getWeights()->getWeightMatrix(0)->getSize() / sizeof(float);
	// 				if (nextMaxWeightIndex > nextNumWeights) {
	// 					nextMaxWeightIndex = nextNumWeights;
	// 				}
					
	// 				gpuErrchk(hipFree(next_weights));
	// 				gpuErrchk(hipMalloc((void **) &next_weights, nextMaxWeightIndex * sizeof(float)));
	// 				gpuErrchk(hipMemcpyAsync(next_weights, currentLayer->getNext()->getWeights()->getWeightMatrix(0)->getArr(), nextMaxWeightIndex * sizeof(float), hipMemcpyHostToDevice));
	// 				currentWeightMatrixIndex = 1;
	// 				numWeightsMatrixesLeft = std::ceil((float)nextNumWeights / nextMaxWeightIndex) - 1;
	// 				weightsAddedLastSet = nextMaxWeightIndex;
	// 			}
	// 			weightsFinished = true;
	// 		}
			
	// 		long long helper = 0;

	// 		do {
	// 			if (numOutputsRemaining > 0) {
	// 				// std::cout << "inside22\n";
	// 				// std::cout << "numBlocks: " << numBlocks << '\n';
	// 				// std::cout << "numOutputs: " << numOutputs << '\n';
	// 				// std::cout << "numPerThread: " << numPerThread << '\n';
	// 				// std::cout << "weightsInCurrentKernelRun: " << weightsInCurrentKernelRun << "\n";
	// 				// std::cout << "numOutputsRemaining: " << numOutputsRemaining << '\n';
	// 				// std::cout << "helper: " << helper << '\n';
	// 				// std::cout << "weightsUsed: " << weightsUsed << "\n";
	// 				// std::cout << "numWeightsMatrixesLeft: " << numWeightsMatrixesLeft << "\n";
	// 				// std::cout << "weightsAddedLastSet: " << weightsAddedLastSet << "\n";
	// 				// std::cout << "startingOutputID: " << startingOutputID << "\n\n";
					
	// 				if (numOutputsRemaining - numBlocks < 0) {
	// 					numBlocks = numOutputsRemaining;
	// 				}

	// 				artificialIntelligence::classes::calculateAndUpdateLayerGPUConv<<< numBlocks, numThreads, sharedSize, stream1 >>>(current_input, current_weights, current_output, numBlocks, numOutputs, numPerThread, weightsInCurrentKernelRun, helper, weightsUsed, startingOutputID);
	// 				outputIndex += numBlocks;
	// 				numOutputsRemaining -= numBlocks;
	// 			}
	// 			startingOutputID += numBlocks;
	// 			helper += numBlocks;

	// 		} while (numOutputsRemaining > 0);
	// 		gpuErrchk(hipDeviceSynchronize());
			
	// 		startingOutputID = nextOutputID % numOutputs;
	// 		nextOutputID += weightsInCurrentKernelRun % numOutputs;
	// 		numOutputsRemaining = numOutputs;

	// 		weightsUsed += weightsInCurrentKernelRun;
	// 		numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs;
	// 		weightsInCurrentKernelRun = weightsAddedLastSet;

	// 		float* temp = current_weights;
	// 		current_weights = next_weights;
	// 		next_weights = temp;

	// 	} while (!weightsFinished);
		
	// 	gpuErrchk(hipMemcpy(output, current_output, numOutputs * sizeof(float), hipMemcpyDeviceToHost));
	// 	// printArr(currentWeight->getWeightMatrix(currentWeightMatrixIndex - 1)->getArr(), 10);
	// 	Matrix3D* bias = currentLayer->getBias();
	// 	currentLayer = currentLayer->getNext();
	// 	currentLayerMatrix = currentLayer->getLayerMatrix();
	// 	currentWeight = currentLayer->getWeights();
	// 	numInputs = currentLayerMatrix->getSize() / sizeof(float);

	// 	if (currentLayer->getNext() != nullptr) {
	// 		output = currentLayer->getNext()->getLayerMatrix()->getArr();
	// 		numOutputs = currentLayer->getNext()->getLayerMatrix()->getSize() / sizeof(float);
	// 		numWeights = numInputs * numOutputs;
	// 		maxWeightIndex = currentLayer->getWeights()->getWeightMatrix()->getSize();
	// 		numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs;
	// 		numThreads = 512; // arbitrary
	// 		numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
	// 		output = currentLayer->getNext()->getLayerMatrix()->getArr();
	// 		gpuErrchk(hipFree(next_weights));
	// 		gpuErrchk(hipMalloc((void **) &next_weights, maxWeightIndex * sizeof(float)));
	// 		gpuErrchk(hipFree(current_output));
	// 		gpuErrchk(hipMalloc((void **) &current_output, numOutputs * sizeof(float)));
	// 		gpuErrchk(hipMemset(current_output, 0b00000000, numOutputs * sizeof(float))); 
	// 	}
		
	// 	*currentLayer->getLayerMatrix() += bias;
	// 	sigmoid(currentLayer->getLayerMatrix(), false);
	// 	gpuErrchk(hipFree(current_input));
	// 	gpuErrchk(hipMalloc((void **) &current_input, currentLayerMatrix->getSize()));
	// 	input = currentLayerMatrix->getArr();
	// 	gpuErrchk(hipMemcpy(current_input, input, currentLayerMatrix->getSize(), hipMemcpyHostToDevice));

	// 	debugCounter++;
	// }
	// gpuErrchk(hipFree(current_input));	
	// gpuErrchk(hipFree(current_output));
	// gpuErrchk(hipFree(current_weights));	
	// gpuErrchk(hipFree(next_weights));
	// gpuErrchk(hipStreamDestroy(stream1));
	// gpuErrchk(hipStreamDestroy(stream2));
}

__global__ void artificialIntelligence::classes::calculateAndUpdateLayerGPUConv(float* nodeValues, float* weights, float* output, int inputSize, int outputSize, int numPerThread, long long maxWeightIndex, long long helperIndex, long long startingWeight, int startingOutputId) {
	// extern __shared__ float sdata[];
	// unsigned int tid = threadIdx.x;
	// unsigned long long outputNodeId = (blockIdx.x + startingOutputId) % outputSize;
	// unsigned int numThreads = blockDim.x;
	// unsigned long long weightIndex = tid * outputSize + blockIdx.x + helperIndex;
	// unsigned long long inputNodeId = 0;
	// unsigned int gridSize = numThreads*outputSize;
	// sdata[tid] = 0;

	// while (weightIndex < maxWeightIndex) {
	// 	inputNodeId = (weightIndex + startingWeight) / outputSize;
	// 	sdata[tid] += nodeValues[inputNodeId] * weights[weightIndex];
	// 	weightIndex += gridSize;
	// }

	// __syncthreads();

	// for (unsigned int s=numThreads/2; s>0; s>>=1) {
	// 	if (tid < s) {
	// 		sdata[tid] += sdata[tid + s];
	// 	}
	// 	__syncthreads();
	// }
	
	// if (tid == 0) {
	// 	output[outputNodeId] += sdata[0];
	// }
}

Matrix3D* ConvLayer::calculateErrorCPU (Matrix3D* delta) {
	Matrix3D* currentLayerMatrix = this->getLayerMatrix();
	Matrix3D* error = new Matrix3D(currentLayerMatrix->getLength(), currentLayerMatrix->getWidth(), currentLayerMatrix->getHeight());
	// for (int l = 0; l < currentLayerMatrix->getLength(); l++) {
	// 	for (int w = 0; w < currentLayerMatrix->getWidth(); w++) {
	// 		for (int h = 0; h < currentLayerMatrix->getHeight(); h++) {
	// 			Matrix3D* outputMatrix = this->getNext()->getLayerMatrix();
	// 			Matrix3D* weightedMatrix = new Matrix3D (delta->getLength(), delta->getWidth(), delta->getHeight());
	// 			for (int l2 = 0; l2 < outputMatrix->getLength(); l2++) {
	// 				for (int w2 = 0; w2 < outputMatrix->getWidth(); w2++) {
	// 					for (int h2 = 0; h2 < outputMatrix->getHeight(); h2++) {
	// 						weightedMatrix->insert(*this->getWeights()->getData(l, w, h, l2, w2, h2) * *delta->getData(l2, w2, h2), l2, w2, h2);
	// 					}
	// 				}
	// 			}
	// 			error->insert(weightedMatrix->sum(), l, w, h);
	// 			delete weightedMatrix;
	// 		}
	// 	}
	// }
	return error;
}

Matrix3D* ConvLayer::calculateErrorGPU (Matrix3D* delta) {
	ConvLayer* currentLayer = this;
	Matrix3D* currentLayerMatrix = currentLayer->getLayerMatrix();

	// long long numInputs = currentLayerMatrix->getSize() / sizeof(float);
	// long long numOutputs = currentLayer->getNext()->getLayerMatrix()->getSize() / sizeof(float);
	// long long numWeights = numInputs * numOutputs;
	// long long numInputsRemaining = numInputs;
	// long long inputIndex = 0;
	// long long numBlocks = numInputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numInputs; 
	// long long numThreads = 512;
	// long long maxWeightIndex = numBlocks * numOutputs;
	// long long numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
	// long long sharedSize = numThreads * sizeof(float); 
	// if (maxWeightIndex > numWeights) {
	// 	maxWeightIndex = numWeights;
	// }
	Matrix3D* errorMatrix = new Matrix3D(currentLayer->getLayerMatrix()->getLength(), currentLayer->getLayerMatrix()->getWidth(), currentLayer->getLayerMatrix()->getHeight());
	// float* error = errorMatrix->getArr();
	// float* current_error;
	// float* current_delta;
	// gpuErrchk(hipMalloc((void **) &current_error, errorMatrix->getSize()));
	// gpuErrchk(hipMalloc((void **) &current_delta, delta->getSize()));
	// gpuErrchk(hipMemcpy(current_error, error, errorMatrix->getSize(), hipMemcpyHostToDevice));
	// gpuErrchk(hipMemcpy(current_delta, delta->getArr(), delta->getSize(), hipMemcpyHostToDevice));

	// hipStream_t stream1, stream2;
	// hipStreamCreate ( &stream1); 
	// hipStreamCreate ( &stream2); 
	
	// ConvWeight* currentWeight = currentLayer->getWeights();
	// long long matrixSize = currentWeight->getWeightMatrix()->getSize() / sizeof(float);
	// long long currentWeightMatrixIndex = 0;
	// long long weightsAddedLastSet = 0;
	// long long weightsInCurrentKernelRun = 0;

	// // std::cout << "Number of threads: " << numThreads << '\n';
	// // std::cout << "Number of blocks: " << numBlocks << '\n';
	// // std::cout << "Number per thread: " << numPerThread << '\n';
	// // std::cout << "Number of bytes for shared storage: " << sharedSize << "\n";
	// // std::cout << "Max array index: " << maxWeightIndex << "\n";
	// // std::cout << "Max byte index: " << maxWeightIndex * sizeof(float) << "\n";
	// // std::cout << "numLeftToAdd: " <<  numLeftToAdd << "\n";

	// float* current_weights;
	// float* next_weights;

	// gpuErrchk(hipMalloc((void **) &current_weights, maxWeightIndex * sizeof(float)));
	// gpuErrchk(hipMalloc((void **) &next_weights, maxWeightIndex * sizeof(float)));

	// int weightsInCurrentMatrix = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
	// int weightsInConvWeight = currentWeight->getSize();

	// int numberOfWeightsToAdd = numBlocks * numOutputs;
	// int toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;

	// int amountAdded = 0;
	// int weightsAdded = 0;

	// while (numberOfWeightsToAdd > 0) {
	// 	toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;
	// 	// std::cout << "\ntoAdd: " <<  toAdd << "\n";
	// 	// std::cout << "numberOfWeightsToAdd: " <<  numberOfWeightsToAdd << "\n";
	// 	// std::cout << "weightsInCurrentMatrix: " <<  weightsInCurrentMatrix << "\n";
	// 	// std::cout << "currentWeightMatrixIndex: " <<  currentWeightMatrixIndex << "\n";
	// 	gpuErrchk(hipMemcpy(&current_weights[weightsAdded], &currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded], toAdd * sizeof(float), hipMemcpyHostToDevice));
	// 	if (toAdd == weightsInCurrentMatrix) {
	// 		currentWeightMatrixIndex++;
	// 		// std::cout << "inside1\n";
	// 		numberOfWeightsToAdd -= toAdd;
	// 		amountAdded = 0;
	// 		weightsAdded += toAdd;
	// 		if (weightsAdded < numWeights) {
	// 			weightsInCurrentMatrix = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
	// 		}
	// 	} else {
	// 		amountAdded = toAdd;
	// 		numberOfWeightsToAdd = 0;
	// 		weightsInCurrentMatrix -= toAdd;
	// 		weightsAdded += toAdd;
	// 	}
	// }
	// weightsInCurrentKernelRun = weightsAdded;
	// weightsAddedLastSet = weightsAdded;
	
	// int startingInputID = 0;
	// int numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;

	// numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;
	// inputIndex = 0;
	// startingInputID = 0;
	// numInputsRemaining = numInputs;
	// bool weightsFinished = false;
	// long long weightsUsed = 0;
	// do {
	// 	if (numInputsRemaining > 0) {
	// 		numBlocks = (weightsUsed + weightsInCurrentKernelRun) * numInputs / numWeights - weightsUsed * numInputs / numWeights;
	// 		// std::cout << "inside22\n";
	// 		// std::cout << "numBlocks: " << numBlocks << '\n';
	// 		// std::cout << "numInputs: " << numInputs << '\n';
	// 		// std::cout << "numPerThread: " << numPerThread << '\n';
	// 		// std::cout << "weightsInCurrentKernelRun: " << weightsInCurrentKernelRun << "\n";
	// 		// std::cout << "numInputsRemaining: " << numInputsRemaining << '\n';
	// 		// std::cout << "weightsUsed: " << weightsUsed << "\n";
	// 		// std::cout << "startingInputID: " << startingInputID << "\n\n";

	// 		if (numInputsRemaining - numBlocks < 0) {
	// 			numBlocks = numInputsRemaining;
	// 		}
			
	// 		artificialIntelligence::classes::calculateErrorConv<<< numBlocks, numThreads, sharedSize, stream1 >>>(current_weights, current_delta, current_error, numInputs, numOutputs, numPerThread, weightsInCurrentKernelRun, numWeights, weightsUsed, startingInputID);
	// 		inputIndex += numBlocks;
	// 		numInputsRemaining -= numBlocks;
	// 	}
	// 	gpuErrchk(hipDeviceSynchronize());
	// 	startingInputID += numBlocks;

	// 	weightsUsed += weightsInCurrentKernelRun;
	// 	weightsInCurrentKernelRun = weightsAddedLastSet;
	// 	if (numWeights - weightsAdded > 0) {
	// 		// std::cout << "here\n";
	// 		// exit(0);
	// 		// asynchronously add the next set of weights
	// 		numBlocks = numInputsRemaining > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numInputsRemaining;
	// 		numberOfWeightsToAdd = numBlocks * numOutputs;
	// 		toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;
	// 		amountAdded = weightsAdded % (WEIGHT_MAX_SIZE);
	// 		int weightCounter = 0;
	// 		if (weightsInCurrentMatrix > 0) {
	// 			weightsInCurrentKernelRun = numberOfWeightsToAdd;
	// 			while (numberOfWeightsToAdd > 0) {
	// 				toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;
	// 				// std::cout << "currentWeightMatrixIndex: " << currentWeightMatrixIndex << "\n";
	// 				// std::cout << "amountAdded: " << amountAdded << "\n";
	// 				// std::cout << "toAdd: " << toAdd << "\n";
	// 				// std::cout << "weightsInCurrentMatrix: " << weightsInCurrentMatrix << "\n";
	// 				// std::cout << "currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded]: " << currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded] << '\n';
	// 				gpuErrchk(hipMemcpyAsync(&next_weights[weightCounter], &currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded], toAdd * sizeof(float), hipMemcpyHostToDevice));
	// 				if (toAdd == weightsInCurrentMatrix) {
	// 					currentWeightMatrixIndex++;
	// 					numberOfWeightsToAdd -= toAdd;
	// 					amountAdded = 0;
	// 					weightsAdded += toAdd;
	// 					weightCounter += toAdd;
	// 					if (weightsAdded < numWeights) {
	// 						weightsInCurrentMatrix = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
	// 					}
	// 				} else {
	// 					numberOfWeightsToAdd = 0;
	// 					weightsInCurrentMatrix -= toAdd;
	// 					weightsAdded += toAdd;
	// 				}
	// 			}
	// 		}
	// 	}
	// 	else {
	// 		weightsFinished = true;
	// 	}

	// 	gpuErrchk(hipDeviceSynchronize());

	// 	float* temp = current_weights;
	// 	current_weights = next_weights;
	// 	next_weights = temp;

	// } while (!weightsFinished);

	// gpuErrchk(hipMemcpy(error, current_error, numInputs * sizeof(float), hipMemcpyDeviceToHost));

	// // :::: FREE ALL ALLOCATED MEMORY :::: //
	// gpuErrchk(hipFree(current_error));	
	// gpuErrchk(hipFree(current_delta));
	// gpuErrchk(hipFree(current_weights));	
	// gpuErrchk(hipFree(next_weights));
	// gpuErrchk(hipStreamDestroy(stream1));
	// gpuErrchk(hipStreamDestroy(stream2));
	return errorMatrix;
}

__global__ void artificialIntelligence::classes::calculateErrorConv(float* weights, float* delta, float* error, int inputSize, int outputSize, int numPerThread, long long maxWeightIndex, long long helperIndex, long long startingWeight, int startingInputID) {
	// extern __shared__ float sdata[];
	// unsigned int tid = threadIdx.x;
	// unsigned int numThreads = blockDim.x;
	// unsigned long long inputNodeId = blockIdx.x + startingInputID;
	// unsigned long long weightIndex = tid + blockIdx.x * outputSize;
	// unsigned int gridSize = numThreads;
	// int weightsToAddStart = outputSize * (blockIdx.x);
	// int weightsToAddEnd = outputSize * (blockIdx.x + 1);

	// sdata[tid] = 0;
	// while (weightIndex >= weightsToAddStart && weightIndex < weightsToAddEnd) {
	// 	sdata[tid] += weights[weightIndex] * delta[(startingWeight + weightIndex) % outputSize];
	// 	weightIndex += gridSize;
	// }

	// __syncthreads();

	// for (unsigned int s=numThreads/2; s>0; s>>=1) {
	// 	if (tid < s) {
	// 		sdata[tid] += sdata[tid + s];
	// 	}
	// 	__syncthreads();
	// }
	
	// if (tid == 0) {
	// 	error[inputNodeId] += sdata[0];
	// }
}

void ConvLayer::updateWeightsCPU (Matrix3D* delta, double learningRate) {
	// Matrix3D* currentLayerMatrix = this->getLayerMatrix();
	// for (int l = 0; l < currentLayerMatrix->getLength(); l++) {
	// 	for (int w = 0; w < currentLayerMatrix->getWidth(); w++) {
	// 		for (int h = 0; h < currentLayerMatrix->getHeight(); h++) {
	// 			float inputValue = *currentLayerMatrix->getData(l, w, h);
	// 			float value = 0;
				
	// 			Matrix3D* weightMatrix = this->getNext()->getLayerMatrix();
	// 			for (int l2 = 0; l2 < weightMatrix->getLength(); l2++) {
	// 				for (int w2 = 0; w2 < weightMatrix->getWidth(); w2++) {
	// 					for (int h2 = 0; h2 < weightMatrix->getHeight(); h2++) {
	// 						value = *this->getWeights()->getData(l, w, h, l2, w2, h2) + inputValue * *delta->getData(l2, w2, h2) * learningRate;
	// 						this->getWeights()->insertData(value, l, w, h, l2, w2, h2);
	// 					}
	// 				}
	// 			}
	// 		}
	// 	}
	// }
}

void ConvLayer::updateWeightsGPU (Matrix3D* delta, double learningRate) {
	// ConvLayer* currentLayer = this;
	// Matrix3D* currentLayerMatrix = currentLayer->getLayerMatrix();

	// long long numInputs = currentLayerMatrix->getSize() / sizeof(float);
	// long long numOutputs = currentLayer->getNext()->getLayerMatrix()->getSize() / sizeof(float);
	// long long numWeights = numInputs * numOutputs;
	// long long inputIndex = 0;
	// long long numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs; 
	// long long numThreads = 512;
	// long long maxWeightIndex = numBlocks * numOutputs;
	// long long numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
	// long long sharedSize = numThreads * sizeof(float); 
	// if (maxWeightIndex > numWeights) {
	// 	maxWeightIndex = numWeights;
	// }
	
	// hipStream_t stream1, stream2;
	// hipStreamCreate ( &stream1); 
	// hipStreamCreate ( &stream2);

	// ConvWeight* currentWeight = currentLayer->getWeights();
	// long long matrixSize = currentWeight->getWeightMatrix()->getSize() / sizeof(float);
	// long long currentWeightMatrixIndex = 0;
	// long long weightsInCurrentKernelRun = 0;
	
	// int weightsInCurrentMatrix = currentWeight->getWeightMatrix(0)->getSize() / sizeof(float);

	// Matrix3D* inputMatrix = currentLayer->getLayerMatrix();
	// float* current_input;
	// float* current_delta;
	// gpuErrchk(hipMalloc((void **) &current_input, inputMatrix->getSize()));
	// gpuErrchk(hipMalloc((void **) &current_delta, delta->getSize()));
	// gpuErrchk(hipMemcpy(current_input, inputMatrix->getArr(), inputMatrix->getSize(), hipMemcpyHostToDevice));
	// gpuErrchk(hipMemcpy(current_delta, delta->getArr(), delta->getSize(), hipMemcpyHostToDevice));
	

	// float* current_weights;
	// float* next_weights;
	// gpuErrchk(hipMalloc((void **) &current_weights, currentWeight->getWeightMatrix(0)->getSize()));
	// gpuErrchk(hipMalloc((void **) &next_weights, currentWeight->getWeightMatrix(0)->getSize()));
	// gpuErrchk(hipMemcpy(current_weights, currentWeight->getWeightMatrix(0)->getArr(), currentWeight->getWeightMatrix(0)->getSize(), hipMemcpyHostToDevice));
	// weightsInCurrentKernelRun = currentWeight->getWeightMatrix(0)->getSize() / sizeof(float);

	// long long weightsUsed = 0;
	// int startingInputId = 0;
	// while ((numWeights - weightsUsed) != 0) {
	// 	// std::cout << "inside22\n";
	// 	// std::cout << "numBlocks: " << numBlocks << '\n';
	// 	// std::cout << "numOutputs: " << numOutputs << '\n';
	// 	// std::cout << "numPerThread: " << numPerThread << '\n';
	// 	// std::cout << "weightsInCurrentKernelRun: " << weightsInCurrentKernelRun << "\n";
	// 	// std::cout << "numOutputsRemaining: " << numOutputsRemaining << '\n';
	// 	// std::cout << "weightsUsed: " << weightsUsed << "\n";
	// 	// std::cout << "startingInputId: " << startingInputId << "\n\n";

	// 	artificialIntelligence::classes::updateWeightsConv<<<numBlocks, numThreads, sharedSize, stream1>>>(current_weights, current_delta, current_input, numInputs, numOutputs, numPerThread, weightsInCurrentKernelRun, numWeights, weightsUsed, startingInputId, learningRate);
	// 	inputIndex += numBlocks;
		
	// 	startingInputId = weightsUsed / numOutputs;

	// 	weightsUsed += currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);

	// 	currentWeightMatrixIndex++;
	// 	if ((numWeights - weightsUsed) != 0) {
	// 		gpuErrchk(hipMemcpyAsync(next_weights, currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr(), currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize(), hipMemcpyHostToDevice));
	// 		weightsInCurrentKernelRun = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
	// 	}

	// 	gpuErrchk(hipDeviceSynchronize());
	// 	gpuErrchk(hipMemcpy(currentWeight->getWeightMatrix(currentWeightMatrixIndex - 1)->getArr(), current_weights, currentWeight->getWeightMatrix(currentWeightMatrixIndex - 1)->getSize(), hipMemcpyDeviceToHost));

	// 	float* temp = current_weights;
	// 	current_weights = next_weights;
	// 	next_weights = temp;
	// }

	// gpuErrchk(hipFree(current_input));
	// gpuErrchk(hipFree(current_delta));
	// gpuErrchk(hipFree(current_weights));	
	// gpuErrchk(hipFree(next_weights));
	// gpuErrchk(hipStreamDestroy(stream1));
	// gpuErrchk(hipStreamDestroy(stream2));
}

__global__ void artificialIntelligence::classes::updateWeightsConv(float* weights, float* delta, float* input, int inputSize, int outputSize, int numPerThread, long long maxWeightIndex, long long helperIndex, long long startingWeight, int startingInputID, double learningRate) {
	// extern __shared__ float sdata[];
	// unsigned int tid = threadIdx.x;
	// unsigned int numThreads = blockDim.x;
	// unsigned long long weightIndex = tid + numThreads * blockIdx.x;
	// unsigned long long outputNodeId = (weightIndex + startingWeight) % outputSize;
	// unsigned long long inputNodeId = (weightIndex + startingWeight) / outputSize;
	// unsigned int gridSize = numThreads * gridDim.x;
	// while (weightIndex < maxWeightIndex) {
	// 	weights[weightIndex] += input[inputNodeId] * delta[outputNodeId] * learningRate;
	// 	weightIndex += gridSize;
	// 	inputNodeId = (weightIndex + startingWeight) / outputSize;
	// 	outputNodeId = (weightIndex + startingWeight) % outputSize;
	// }
}

void ConvLayer::printDetails () {
	std::cout << "Conv Layer :: ";
	this->getLayerMatrix()->printMatrixSize();
	std::cout << "Conv Size :: " << "[" << this->convLength << "x" << this->convWidth << "x" << this->convHeight << "]\n";
	std::cout << "Features :: " << this->features << '\n';
}  


void ConvLayer::toFile (std::ofstream* outputFile) {
	// char* output = new char[sizeof(int) * 6];
   // *outputFile << this->getLayerMatrix()->getLength() << ',' << this->getLayerMatrix()->getWidth() << ',' << this->getLayerMatrix()->getHeight() << '\n';

   // if (this->biasMatrixes[0] == nullptr) {
   //    return;
   // }
   // *outputFile << this->getBias()->getLength() << ',' << this->getBias()->getWidth() << ',' << this->getBias()->getHeight() << '\n';
   // for (int i = 0; i < this->getBias()->getLength(); i++) {
   //    for (int j = 0; j < this->getBias()->getWidth(); j++) {
   //       for (int k = 0; k < this->getBias()->getHeight(); k++) {
   //          *outputFile << *this->getBias()->getData(i, j, k) << ',';
   //       }
   //    }
   // }

   // outputFile->seekp((int) outputFile->tellp() - 1);
   // outputFile->write("\n", 1);

   // if (this->weights[0] == nullptr) {
   //    return;
   // }

   // *outputFile << this->getLayerMatrix()->getLength() << ',' << this->getLayerMatrix()->getWidth() << ',' << this->getLayerMatrix()->getHeight() << ',';
   // *outputFile << this->getBias()->getLength() << ',' << this->getBias()->getWidth() << ',' << this->getBias()->getHeight() << '\n';

	// int currentWeightMatrix = 0;
	// float* weights;
	
	// while (this->getWeights()->getWeightMatrix(currentWeightMatrix) != nullptr) {
	// 	int size = this->getWeights()->getWeightMatrix(currentWeightMatrix)->getSize() + sizeof(float);
	// 	char* output = new char[size];
	// 	char* ptr = output;

	// 	weights = this->getWeights()->getWeightMatrix(currentWeightMatrix)->getArr();
	// 	for (int i = 0, cc = this->getWeights()->getWeightMatrix(currentWeightMatrix)->getSize() / sizeof(float); i < cc; i++) {
	// 		memcpy(ptr, &weights[i], sizeof(float));
	// 		ptr += sizeof(float);
	// 	}
	// 	outputFile->write(output, size);
	// 	currentWeightMatrix++;

	// 	free(output);

	// }

   // outputFile->seekp((int) outputFile->tellp() - 1);
   // outputFile->write("\n", 1);

   // if (this->getNext() == nullptr) {
   //    return;
   // }
   // this->getNext()->toFile(outputFile);
}


LayerBase* ConvLayer::loadFromFile (std::ifstream* inputFile, LayerBase* prev) {
	// std::cout << "Loading layer from file\n";
   ConvLayer* layer = new ConvLayer ();
   // std::string line;
   // getline (*inputFile, line);
   // std::stringstream lineStream;
   // lineStream << line;
   // std::string value;
   // getline(lineStream, value, ',');
	// std::cout << "v1: " << value << '\n';
   // int layerLength = stoi(value);
   // getline(lineStream, value, ',');
	// std::cout << "v2: " << value << '\n';
   // int layerWidth = stoi(value);
   // getline(lineStream, value, ',');
	// std::cout << "v3: " << value << '\n';
   // int layerHeight = stoi(value);
   // Matrix3D* layerMatrix = new Matrix3D (layerLength, layerWidth, layerHeight);
   // layer->layerMatrix = layerMatrix;
   // layer->prev[0] = prev;

   // lineStream.str(std::string());
   // lineStream.clear();
   // getline (*inputFile, line);
   // lineStream << line;

   // if (inputFile->eof()) {
	// 	layer->biasMatrixes = new Matrix3D*[1];
   // 	layer->weights = (WeightBase**) new ConvWeight*[1];
   //    return layer;
   // }

   // getline(lineStream, value, ',');
   // int biasLength = stoi(value);
   // getline(lineStream, value, ',');
   // int biasWidth = stoi(value);
   // getline(lineStream, value, ',');
   // int biasHeight = stoi(value);
   // Matrix3D* biasMatrix = new Matrix3D (biasLength, biasWidth, biasHeight);
   // layer->biasMatrixes[0] = biasMatrix;

   // lineStream.str(std::string());
   // lineStream.clear();
   // getline (*inputFile, line);
   // lineStream << line;
   // for (int i = 0; i < layer->getBias()->getLength(); i++) {
   //    for (int j = 0; j < layer->getBias()->getWidth(); j++) {
   //       for (int k = 0; k < layer->getBias()->getHeight(); k++) {
   //          std::getline(lineStream, value, ',');
   //          layer->getBias()->insert (stod(value), i, j, k);
   //       }
   //    }
   // }

   // getline (*inputFile, line);

   // if (inputFile->eof()) {
	// 	layer->weights = (WeightBase**) new ConvWeight*[1];
   //    return layer;
   // }

   // ConvWeight* weights = new ConvWeight (
   //    layer->getLayerMatrix()->getLength(), 
   //    layer->getLayerMatrix()->getWidth(), 
   //    layer->getLayerMatrix()->getHeight(), 
   //    layer->getBias()->getLength(), 
   //    layer->getBias()->getWidth(), 
   //    layer->getBias()->getHeight(),
	// 	0
   // );

   
   // lineStream.str(std::string());
   // lineStream.clear();
	
	// std::cout << "Inserting weights\n";

	// int currentWeightMatrix = 0;
	// while (weights->getWeightMatrix(currentWeightMatrix) != nullptr) {
	// 	inputFile->read((char*) weights->getWeightMatrix(currentWeightMatrix)->getArr(), weights->getWeightMatrix(currentWeightMatrix)->getSize());
	// 	currentWeightMatrix++;
	// }
	// getline(*inputFile, line);

	// std::cout << "Finished weights\n";

   // layer->weights[0] = weights;
   // layer->next[0] = ConvLayer::loadFromFile (inputFile, layer);

   return layer;
}